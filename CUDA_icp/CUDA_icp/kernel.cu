#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"

#include ""

#include<thrust/device_vector.h>

#include<thrust/sort.h>

#include "cuda_function.h"

#include <math.h>

#include<time.h>

#define  BLOCK_NUM 32
#define  THREAD_NUM 1024

using namespace cuda_function;


__global__ void Distance(float px, float py, float pz, float*qx, float *qy, float *qz, int qsize, float *dist)

{

	int tid = threadIdx.x;

	int bid = blockIdx.x;

		for (int j = tid + bid*THREAD_NUM; j < qsize; j += BLOCK_NUM*THREAD_NUM)
		{
			dist[j] = pow(px - *(qx + j), 2) + pow(py - *(qy + j), 2) + pow(pz - *(qz + j), 2);
			 //printf("%d:  %f\n", j, dist[j]);
			//if (dist[j] - 0 < 0.0000001) { printf("%d: %f\n", j, dist[j]); }
		}
		__syncthreads();
}


void cuda_function::FindCorrespondantPoint(PointSet &P, PointSet &Q, PointSet &X, float &E)
{
	float *qx, *qy, *qz,*dist;
	E = 0;
	hipMalloc((void**)&qx, Q.size * sizeof(float));
	hipMemcpy(qx, Q.x, Q.size * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&qy, Q.size * sizeof(float));
	hipMemcpy(qy, Q.y, Q.size * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&qz, Q.size * sizeof(float));
	hipMemcpy(qz, Q.z, Q.size * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&dist, Q.size * sizeof(float));
	float *distance;
	distance = new float[Q.size];
	/*printf("%f\n", px[0]);
	Distance << <BLOCK_NUM, THREAD_NUM >> > (P.x[0], P.y[0], P.z[0], qx, qy, qz, Q.size, dist);
	hipDeviceSynchronize();
	hipMemcpy(distance, dist, Q.size * sizeof(float), hipMemcpyDeviceToHost);
	printf("%f,%f\n", dist[0], distance[0]);*/
	for (int i = 0; i < P.size; i++)
	{
		//clock_t s=clock();
		//printf("%d:\n", i);
		Distance << <BLOCK_NUM, THREAD_NUM >> > (P.x[i], P.y[i], P.z[i], qx, qy, qz, Q.size, dist);
		hipDeviceSynchronize();
		//printf("calculation cost %f seconds\n", (float)(clock() - s) / CLOCKS_PER_SEC);		
		hipMemcpy(&*distance, dist, Q.size * sizeof(float), hipMemcpyDeviceToHost);

		//printf("memcopy cost %f seconds\n", (float)(clock() - s) / CLOCKS_PER_SEC);
		float mindist = 1000000;
		int pos;
		for (int j = 0; j < Q.size; j++)
		{
			if (mindist > distance[j]) {
				mindist = distance[j];
				pos = j;
			}
		}
		
		//printf("comparation cost %f seconds\n", (float)(clock() - s) / CLOCKS_PER_SEC);
		E += mindist/P.size;// printf("%f, %d\n", mindist,pos);
		//printf("this00 cost %f seconds\n", (float)(clock() - s) / CLOCKS_PER_SEC);
		X.x[i] = Q.x[pos];
		X.y[i] = Q.y[pos];
		X.z[i] = Q.z[pos];
		//printf("this01 cost %f seconds\n", (float)(clock() - s) / CLOCKS_PER_SEC);
		
	}
	hipFree(qx);
	hipFree(qy);
	hipFree(qz);
	hipFree(dist);
}
